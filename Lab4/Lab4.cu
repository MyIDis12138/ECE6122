/*
Author:  Yang Gu
Date last modified: 31/10/2023
Organization: ECE6122 Class

Description:

This program perfroms CUDA-based 2D Random Walk Simulation to compare 
the performance of three different types of CUDA memory allocation:
1. Standard CUDA memory allocation
2. Pinned (page-locked) memory
3. Managed (unified) memory
    
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <chrono>
#include <cassert>
#include <cstring>
#include <cstdlib>

/**
 * CUDA kernel to simulate random walks.
 * @param x - Pointer to the array storing x-coordinates of walkers.
 * @param y - Pointer to the array storing y-coordinates of walkers.
 * @param num_steps - Number of steps for each walker.
 * @param num_walkers - Total number of walkers.
 */
__global__ void randomWalkKernel(int *x, int *y, int num_steps, int num_walkers) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(tid, tid, 0, &state);

    int local_x = 0;
    int local_y = 0;

    for (int i = 0; i < num_steps; ++i) {
        float rnd = hiprand_uniform(&state);
        if (rnd < 0.25f)
            local_x++;
        else if (rnd < 0.5f)
            local_x--;
        else if (rnd < 0.75f)
            local_y++;
        else
            local_y--;

    }

    x[tid] = local_x;
    y[tid] = local_y;
}

/**
 * Calculate the average distance of walkers from the origin.
 * @param x - Array of x-coordinates.
 * @param y - Array of y-coordinates.
 * @param num_walkers - Number of walkers.
 * @return The average distance.
 */
float calculateAverageDistance(int *x, int *y, const int num_walkers) {
    float total_distance = 0.0;
    for (int i = 0; i < num_walkers; ++i) {
        total_distance += sqrtf(x[i] * x[i] + y[i] * y[i]);
    }
    return total_distance / num_walkers;
}

/**
 * Conducts the random walk and prints out results.
 * @param x - Array of x-coordinates.
 * @param y - Array of y-coordinates.
 * @param num_walkers - Number of walkers.
 * @param num_steps - Number of steps for each walker.
 * @param blocksPerGrid - Number of blocks in the CUDA grid.
 * @param blockSize - Size of each block (number of threads).
 * @param memoryType - Type of CUDA memory used (e.g., "Normal", "Pinned", "Managed").
 */
void performRandomWalkAndReport(int *x, int *y, const int num_walkers, const int num_steps, const int blocksPerGrid, const int blockSize, const char* memoryType) {
    auto start = std::chrono::high_resolution_clock::now();
    randomWalkKernel<<<blocksPerGrid, blockSize>>>(x, y, num_steps, num_walkers);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    float avg_distance = calculateAverageDistance(x, y, num_walkers);
    std::cout << memoryType << " CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << duration.count() << std::endl;
    std::cout << "    Average distance from origin: " << avg_distance << std::endl;
}


/**
 * Random walk simulation using standard CUDA memory allocation.
 * @param num_walkers - Number of walkers.
 * @param num_steps - Number of steps for each walker.
 * @param blocksPerGrid - Number of blocks in the CUDA grid.
 * @param blockSize - Size of each block (number of threads).
 */
void normalMemoryAllocation(const int &num_walkers, const int &num_steps, const int &blocksPerGrid, const int &blockSize) {
    int *x, *y;
    hipMalloc(&x, num_walkers * sizeof(int));
    hipMalloc(&y, num_walkers * sizeof(int));

    auto start = std::chrono::high_resolution_clock::now();
    randomWalkKernel<<<blocksPerGrid, blockSize>>>(x, y, num_steps, num_walkers);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    int *host_x = new int[num_walkers];
    int *host_y = new int[num_walkers];
    hipMemcpy(host_x, x, num_walkers * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_y, y, num_walkers * sizeof(int), hipMemcpyDeviceToHost);

    float avg_distance = calculateAverageDistance(host_x, host_y, num_walkers);
    std::cout << "Normal CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << duration.count() << std::endl;
    std::cout << "    Average distance from origin: " << avg_distance << std::endl;

    hipFree(x);
    hipFree(y);
    delete[] host_x;
    delete[] host_y;
}
/**
 * Random walk simulation using pinned (page-locked) memory.
 * @param num_walkers - Number of walkers.
 * @param num_steps - Number of steps for each walker.
 * @param blocksPerGrid - Number of blocks in the CUDA grid.
 * @param blockSize - Size of each block (number of threads).
 */
void pinnedMemoryAllocation(const int &num_walkers, const int &num_steps, const int &blocksPerGrid, const int &blockSize) {
    int *pinned_x, *pinned_y;
    hipHostMalloc(&pinned_x, num_walkers * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&pinned_y, num_walkers * sizeof(int), hipHostMallocDefault);

    performRandomWalkAndReport(pinned_x, pinned_y, num_walkers, num_steps, blocksPerGrid, blockSize, "Pinned");

    hipHostFree(pinned_x);
    hipHostFree(pinned_y);
}

/**
 * Random walk simulation using managed (unified) memory.
 * @param num_walkers - Number of walkers.
 * @param num_steps - Number of steps for each walker.
 * @param blocksPerGrid - Number of blocks in the CUDA grid.
 * @param blockSize - Size of each block (number of threads).
 */
void managedMemoryAllocation(const int &num_walkers, const int &num_steps, const int &blocksPerGrid, const int &blockSize) {
    int *managed_x, *managed_y;
    hipMallocManaged(&managed_x, num_walkers * sizeof(int));
    hipMallocManaged(&managed_y, num_walkers * sizeof(int));

    performRandomWalkAndReport(managed_x, managed_y, num_walkers, num_steps, blocksPerGrid, blockSize, "Managed");

    hipFree(managed_x);
    hipFree(managed_y);
}

int main(int argc, char **argv) {
    int num_walkers = 0;
    int num_steps = 0;

    assert(argc == 5 && "Invalid number of arguments. Usage: Lab4 -W <number of walkers> -I <number of steps>");


    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-W") == 0) {
            num_walkers = atoi(argv[++i]);
        } else if (strcmp(argv[i], "-I") == 0) {
            num_steps = atoi(argv[++i]);
        }
    }

    assert(num_walkers > 0 && "Number of walkers must be greater than 0");
    assert(num_steps > 0 && "Number of steps must be greater than 0");

    std::cout << "Lab4 -W " << num_walkers << " -I " << num_steps << std::endl;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int blockSize = deviceProp.maxThreadsPerBlock / 4;
    int blocksPerGrid = (num_walkers + blockSize - 1) / blockSize;

    normalMemoryAllocation(num_walkers, num_steps, blocksPerGrid, blockSize);

    pinnedMemoryAllocation(num_walkers, num_steps, blocksPerGrid, blockSize);

    managedMemoryAllocation(num_walkers, num_steps, blocksPerGrid, blockSize);

    std::cout << "Bye" << std::endl;
    return 0;
}
