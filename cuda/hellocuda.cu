#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <assert.h>

#define N 10000000

#define MAX_ERR 1e-6

void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

__global__ void vector_CUDAadd(float *out, float *a, float *b, int n)
{ 
//    int index = threadIdx.x;
//    int stride = blockDim.x;
    int tid = threadIdx.x * blockDim.x + blockDim.x;

    if (tid < N){
        out[tid] = a[tid] + b[tid];
    }
//    for (int i = 0; i < n; i += stride){
//        out[i] = a[i] + b[i];
//    }
}

int main(){
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out;
    float size = N * sizeof(float);

    // Allocate memory
    a   = (float*)malloc(size);
    b   = (float*)malloc(size);
    out = (float*)malloc(size);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_out, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);

    vector_CUDAadd<<<grid_size, block_size>>>(d_out, d_a, d_b, N);

//    cudaMemcpy(out, d_out, size, cudaMemcpyDeviceToHost);

//    for(int i = 0; i < N; i++){
//        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
//    }

    free(a); free(b); free(out);
    hipFree(d_a); hipFree(d_b); hipFree(d_out);

}

